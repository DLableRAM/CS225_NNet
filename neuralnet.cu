#include "defs.cuh"

neuralnet::neuralnet(int ins, int ops, int hls, int hlc, std::string n) {
  name = n;
  inputSize = ins;
  outputSize = ops;
  hiddenLayerSize = hls;
  hiddenLayerCount = hlc;
  input = new float[inputSize];
  output = new float[hiddenLayerSize*hiddenLayerCount + outputSize];
  wmatrix = new float[hiddenLayerSize*hiddenLayerSize*hiddenLayerCount + inputSize*hiddenLayerSize + outputSize*hiddenLayerSize];
  bias = new float[hiddenLayerSize*hiddenLayerCount + outputSize];
}

neuralnet::~neuralnet() {
  delete [] input;
  delete [] output;
  delete [] wmatrix;
  delete [] bias;
  hipFree(device_input);
  hipFree(device_output);
  hipFree(device_wmatrix);
  hipFree(device_bias);
}

void neuralnet::loadNet() {
  //TODO: Add a safeguard to prevent loading models multiple times.
  int inputDataSize = inputSize*sizeof(float);
  int outputDataSize = (hiddenLayerSize*hiddenLayerCount + outputSize)*sizeof(float);
  int wmatrixDataSize = (hiddenLayerSize*hiddenLayerSize*hiddenLayerCount + inputSize*hiddenLayerSize + outputSize*hiddenLayerSize)*sizeof(float);
  int biasDataSize = (hiddenLayerSize*hiddenLayerCount + outputSize)*sizeof(float);

  //TODO: Add malloc error checking.
  hipMalloc(&device_input, inputDataSize);
  hipMalloc(&device_output, outputDataSize);
  hipMalloc(&device_wmatrix, wmatrixDataSize);
  hipMalloc(&device_bias, biasDataSize);

  hipMemcpy(device_input, input, inputDataSize, hipMemcpyHostToDevice);
  hipMemcpy(device_output, output, outputDataSize, hipMemcpyHostToDevice);
  hipMemcpy(device_wmatrix, wmatrix, wmatrixDataSize, hipMemcpyHostToDevice);
  hipMemcpy(device_bias, bias, biasDataSize, hipMemcpyHostToDevice);
}

float* neuralnet::getInput() {
  //I don't think I should hipMemcpy here?
  return input;
}

float* neuralnet::getOutput() {
  //Pulls output from vram
  int outputDataSize = (hiddenLayerSize*hiddenLayerCount + outputSize)*sizeof(float);
  hipMemcpy(device_output, output, outputDataSize, hipMemcpyDeviceToHost);
  return output;
}
