#include "defs.cuh"

neuralnet::neuralnet(int ins, int ops, int hls, int hlc, std::string n) {
  name = n;
  inputSize = ins;
  outputSize = ops;
  hiddenLayerSize = hls;
  hiddenLayerCount = hlc;
  input = new float[inputSize];
  output = new float[hiddenLayerSize*hiddenLayerCount + outputSize];
  wmatrix = new float[hiddenLayerSize*hiddenLayerSize*hiddenLayerCount + inputSize*hiddenLayerSize + outputSize*hiddenLayerSize];
  bias = new float[hiddenLayerSize*hiddenLayerCount + outputSize];
}

neuralnet::~neuralnet() {
  delete [] input;
  delete [] output;
  delete [] wmatrix;
  delete [] bias;
  hipFree(device_input);
  hipFree(device_output);
  hipFree(device_wmatrix);
  hipFree(device_bias);
}

void neuralnet::loadNet() {
  //TODO: Add a safeguard to prevent loading models multiple times.
  int inputDataSize = inputSize*sizeof(float);
  int outputDataSize = (hiddenLayerSize*hiddenLayerCount + outputSize)*sizeof(float);
  int wmatrixDataSize = (hiddenLayerSize*hiddenLayerSize*hiddenLayerCount + inputSize*hiddenLayerSize + outputSize*hiddenLayerSize)*sizeof(float);
  int biasDataSize = (hiddenLayerSize*hiddenLayerCount + outputSize)*sizeof(float);

  hipMalloc(&device_input, inputDataSize);
  hipMalloc(&device_output, outputDataSize);
  hipMalloc(&device_wmatrix, wmatrixDataSize);
  hipMalloc(&device_bias, biasDataSize);
  
  if ((device_input == NULL) || (device_output == NULL) || (device_wmatrix == NULL) || (device_bias == NULL)) {
    errmsg = "Failed to allocate memory in VRAM.";
    throw(errmsg);
  }

  hipMemcpy(device_input, input, inputDataSize, hipMemcpyHostToDevice);
  hipMemcpy(device_output, output, outputDataSize, hipMemcpyHostToDevice);
  hipMemcpy(device_wmatrix, wmatrix, wmatrixDataSize, hipMemcpyHostToDevice);
  hipMemcpy(device_bias, bias, biasDataSize, hipMemcpyHostToDevice);
}

void neuralnet::getOutput(float* out) {
  for (int i = 0; i < outputSize; ++i) {
    out[i] = output[i];
  }
}

void neuralnet::setInput(float* in) {
  for (int i = 0; i < inputSize; ++i) {
    input[i] = in[i];
  }
  //write input to vram
  int inputDataSize = inputSize*sizeof(float);
  hipMemcpy(device_input, input, inputDataSize, hipMemcpyHostToDevice);
}

void neuralnet::infer() {
  //Copy input to vram
  int inputDataSize = inputSize*sizeof(float);
  hipMemcpy(device_input, input, inputDataSize, hipMemcpyHostToDevice);
  //call gpu kernel
  inference<<<numBlocks, numThreads>>>(device_input, inputSize, device_output, outputSize, device_wmatrix, device_bias, hiddenLayerCount, hiddenLayerSize);
  hipDeviceSynchronize();
  int outputDataSize = (hiddenLayerSize*hiddenLayerCount + outputSize)*sizeof(float);
  hipMemcpy(device_output, output, outputDataSize, hipMemcpyDeviceToHost);
}

void neuralnet::trn(std::string directory, float lr, int epochs) {
  //load dataset from files
  std::ifstream fileread;
  int filecount;
  float predictionFiles[filecount*outputSize];
  float inputFiles[filecount*inputSize];
  //begin training loop
  for (epochs = epochs; epochs > 0; --epochs) {
  for (int j = 0; j < filecount; ++j) {
    //load input to vram

    //inference
    //setInput(in);
    infer();
    //get error
    float er;
    float result[outputSize];
    float prediction[outputSize];
    float sum;
    for (int i = 0; i < outputSize; ++i) {
      result[i] = output[hiddenLayerCount*hiddenLayerSize + i];
    }

    for (int i = 0; i < outputSize; ++i) {
      sum += (result[i] - prediction[i]);
    }
    er = (2.0/outputSize)*sum;
    //call kernel
    train<<<numBlocks, numThreads>>>(device_input, lr, device_output, inputSize, er, outputSize, device_wmatrix, device_bias, hiddenLayerCount, hiddenLayerSize);
  }
  }
}

//operator overloading
std::ostream& operator<< (std::ostream& os, const neuralnet& n) {
  os << n.name;
  return os;
}

